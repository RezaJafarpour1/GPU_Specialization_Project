#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdint>
#include <string>
#include <vector>
#include "histeq.hpp"
#include "cuda_utils.hpp"

namespace {
constexpr int TPB = 256; // threads per block

__global__ void kernel_histogram(const uint8_t* __restrict__ in,
                                 int n, unsigned int* __restrict__ g_hist)
{
    __shared__ unsigned int s_hist[256];
    // init shared hist
    for (int i = threadIdx.x; i < 256; i += blockDim.x) s_hist[i] = 0;
    __syncthreads();

    // grid-stride loop over pixels
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    while (idx < n) {
        atomicAdd(&s_hist[in[idx]], 1u);
        idx += stride;
    }
    __syncthreads();

    // reduce to global
    for (int i = threadIdx.x; i < 256; i += blockDim.x) {
        atomicAdd(&g_hist[i], s_hist[i]);
    }
}

__constant__ uint8_t d_lut[256];

__global__ void kernel_apply_lut(const uint8_t* __restrict__ in,
                                 uint8_t* __restrict__ out, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) out[idx] = d_lut[in[idx]];
}
} // namespace

std::string histeq_cuda(const ImageU8& in, ImageU8& out, float* elapsed_ms)
{
    if (in.w <= 0 || in.h <= 0 || (int)in.data.size() != in.w * in.h)
        return "invalid input image";

    out.w = in.w; out.h = in.h; out.data.resize(in.w * in.h);

    const int n = in.w * in.h;
    uint8_t *d_in = nullptr, *d_out = nullptr;
    unsigned int *d_hist = nullptr;

    CUDA_CHECK(hipMalloc(&d_in,  n * sizeof(uint8_t)));
    CUDA_CHECK(hipMalloc(&d_out, n * sizeof(uint8_t)));
    CUDA_CHECK(hipMalloc(&d_hist, 256 * sizeof(unsigned int)));
    CUDA_CHECK(hipMemset(d_hist, 0, 256 * sizeof(unsigned int)));
    CUDA_CHECK(hipMemcpy(d_in, in.data.data(), n * sizeof(uint8_t), hipMemcpyHostToDevice));

    hipEvent_t evs, eve;
    if (elapsed_ms) { hipEventCreate(&evs); hipEventCreate(&eve); hipEventRecord(evs); }

    // Launch histogram
    int blocks = (n + TPB - 1) / TPB;
    blocks = min(blocks, 1024); // cap blocks to keep atomics reasonable
    kernel_histogram<<<blocks, TPB>>>(d_in, n, d_hist);
    auto st = hipGetLastError();
    if (st != hipSuccess) {
        hipFree(d_in); hipFree(d_out); hipFree(d_hist);
        return std::string("histogram launch failed: ") + hipGetErrorString(st);
    }
    CUDA_CHECK(hipDeviceSynchronize());

    // Bring histogram to host, compute CDF & LUT on host
    std::vector<unsigned int> h_hist(256);
    CUDA_CHECK(hipMemcpy(h_hist.data(), d_hist, 256 * sizeof(unsigned int), hipMemcpyDeviceToHost));

    // Compute CDF
    std::vector<unsigned int> cdf(256);
    unsigned int cum = 0; unsigned int cdf_min = 0;
    for (int i = 0; i < 256; ++i) {
        cum += h_hist[i];
        cdf[i] = cum;
        if (cdf_min == 0 && cum != 0) cdf_min = cum;
    }
    std::vector<uint8_t> h_lut(256);
    const unsigned int total = cdf.back();
    if (total == 0 || cdf_min == total) {
        // All pixels same or empty—identity LUT
        for (int i = 0; i < 256; ++i) h_lut[i] = static_cast<uint8_t>(i);
    } else {
        for (int i = 0; i < 256; ++i) {
            float num = float(cdf[i] - cdf_min);
            float den = float(total - cdf_min);
            int v = int(255.0f * (num / den) + 0.5f);
            if (v < 0) v = 0; if (v > 255) v = 255;
            h_lut[i] = static_cast<uint8_t>(v);
        }
    }

    // Upload LUT to constant memory and apply
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_lut), h_lut.data(), 256 * sizeof(uint8_t)));
    kernel_apply_lut<<<(n + TPB - 1) / TPB, TPB>>>(d_in, d_out, n);
    st = hipGetLastError();
    if (st != hipSuccess) {
        hipFree(d_in); hipFree(d_out); hipFree(d_hist);
        return std::string("apply LUT launch failed: ") + hipGetErrorString(st);
    }
    CUDA_CHECK(hipDeviceSynchronize());

    if (elapsed_ms) {
        hipEventRecord(eve); hipEventSynchronize(eve);
        hipEventElapsedTime(elapsed_ms, evs, eve);
        hipEventDestroy(evs); hipEventDestroy(eve);
    }

    CUDA_CHECK(hipMemcpy(out.data.data(), d_out, n * sizeof(uint8_t), hipMemcpyDeviceToHost));
    hipFree(d_in); hipFree(d_out); hipFree(d_hist);
    return {};
}
