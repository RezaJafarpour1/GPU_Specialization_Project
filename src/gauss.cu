#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdint>
#include <string>
#include "gauss.hpp"
#include "cuda_utils.hpp"

namespace {
constexpr int BW = 16;
constexpr int BH = 16;

// Clamp helpers
__device__ inline uint8_t clamp_read_u8(const uint8_t* src, int w, int h, int x, int y) {
    if (x < 0) x = 0; else if (x >= w) x = w - 1;
    if (y < 0) y = 0; else if (y >= h) y = h - 1;
    return src[y * w + x];
}
__device__ inline uint16_t clamp_read_u16(const uint16_t* src, int w, int h, int x, int y) {
    if (x < 0) x = 0; else if (x >= w) x = w - 1;
    if (y < 0) y = 0; else if (y >= h) y = h - 1;
    return src[y * w + x];
}

// Horizontal pass (5-tap): uint8 -> uint16, weights [1 4 6 4 1] (sum = 16)
__global__ void hpass_gauss5(const uint8_t* __restrict__ in,
                             uint16_t* __restrict__ tmp,
                             int w, int h)
{
    __shared__ uint8_t tile[BH][BW + 4]; // halo of 2 (left/right)
    const int tx = threadIdx.x, ty = threadIdx.y;
    const int x  = blockIdx.x * BW + tx;
    const int y  = blockIdx.y * BH + ty;

    // Center
    uint8_t c = (x < w && y < h) ? in[y * w + x] : 0;
    tile[ty][tx + 2] = c;

    // Halos
    if (tx < 2) {
        tile[ty][tx]             = clamp_read_u8(in, w, h, x - 2 + tx, y);   // -2, -1
        tile[ty][tx + BW + 2]    = clamp_read_u8(in, w, h, x + tx + 1, y);   // +1, +2
    }
    __syncthreads();

    if (x >= w || y >= h) return;

    uint16_t a = tile[ty][tx    ];
    uint16_t b = tile[ty][tx + 1];
    uint16_t c0= tile[ty][tx + 2];
    uint16_t d = tile[ty][tx + 3];
    uint16_t e = tile[ty][tx + 4];

    // (a*1 + b*4 + c*6 + d*4 + e*1), still scaled by 16
    uint16_t sum5 = a + (b<<2) + (c0*6) + (d<<2) + e;
    tmp[y * w + x] = sum5;
}

// Vertical pass (5-tap): uint16 -> uint8, weights [1 4 6 4 1], final divide by 256 (16*16)
__global__ void vpass_gauss5(const uint16_t* __restrict__ tmp,
                             uint8_t* __restrict__ out,
                             int w, int h)
{
    __shared__ uint16_t tile[BH + 4][BW]; // halo of 2 (top/bottom)
    const int tx = threadIdx.x, ty = threadIdx.y;
    const int x  = blockIdx.x * BW + tx;
    const int y  = blockIdx.y * BH + ty;

    // Center
    uint16_t c = (x < w && y < h) ? tmp[y * w + x] : 0;
    tile[ty + 2][tx] = c;

    // Halos
    if (ty < 2) {
        tile[ty][tx]             = clamp_read_u16(tmp, w, h, x, y - 2 + ty); // -2, -1
        tile[ty + BH + 2][tx]    = clamp_read_u16(tmp, w, h, x, y + ty + 1); // +1, +2
    }
    __syncthreads();

    if (x >= w || y >= h) return;

    uint32_t a = tile[ty    ][tx];
    uint32_t b = tile[ty + 1][tx];
    uint32_t c0= tile[ty + 2][tx];
    uint32_t d = tile[ty + 3][tx];
    uint32_t e = tile[ty + 4][tx];

    // sum_v = (a*1 + b*4 + c*6 + d*4 + e*1); both passes combined => divide by 256
    uint32_t sum_v = a + (b<<2) + (c0*6) + (d<<2) + e;
    uint32_t v = sum_v >> 8; // /256
    if (v > 255u) v = 255u;
    out[y * w + x] = static_cast<uint8_t>(v);
}
} // namespace

std::string gauss5_cuda(const ImageU8& in, ImageU8& out, float* elapsed_ms)
{
    if (in.w <= 0 || in.h <= 0 || (int)in.data.size() != in.w * in.h)
        return "invalid input image";

    out.w = in.w; out.h = in.h; out.data.resize(in.w * in.h);

    uint8_t  *d_in  = nullptr, *d_out = nullptr;
    uint16_t *d_tmp = nullptr;
    size_t npx   = static_cast<size_t>(in.w) * in.h;
    size_t bytes_u8  = npx * sizeof(uint8_t);
    size_t bytes_u16 = npx * sizeof(uint16_t);

    CUDA_CHECK(hipMalloc(&d_in,  bytes_u8));
    CUDA_CHECK(hipMalloc(&d_out, bytes_u8));
    CUDA_CHECK(hipMalloc(&d_tmp, bytes_u16));
    CUDA_CHECK(hipMemcpy(d_in, in.data.data(), bytes_u8, hipMemcpyHostToDevice));

    dim3 block(BW, BH);
    dim3 grid((in.w + BW - 1) / BW, (in.h + BH - 1) / BH);

    hipEvent_t evs, eve;
    if (elapsed_ms) { hipEventCreate(&evs); hipEventCreate(&eve); hipEventRecord(evs); }

    hpass_gauss5<<<grid, block>>>(d_in, d_tmp, in.w, in.h);
    auto st = hipGetLastError();
    if (st != hipSuccess) {
        hipFree(d_in); hipFree(d_out); hipFree(d_tmp);
        return std::string("gauss hpass launch failed: ") + hipGetErrorString(st);
    }

    vpass_gauss5<<<grid, block>>>(d_tmp, d_out, in.w, in.h);
    st = hipGetLastError();
    if (st != hipSuccess) {
        hipFree(d_in); hipFree(d_out); hipFree(d_tmp);
        return std::string("gauss vpass launch failed: ") + hipGetErrorString(st);
    }

    CUDA_CHECK(hipDeviceSynchronize());

    if (elapsed_ms) {
        hipEventRecord(eve); hipEventSynchronize(eve);
        hipEventElapsedTime(elapsed_ms, evs, eve);
        hipEventDestroy(evs); hipEventDestroy(eve);
    }

    CUDA_CHECK(hipMemcpy(out.data.data(), d_out, bytes_u8, hipMemcpyDeviceToHost));
    hipFree(d_in); hipFree(d_out); hipFree(d_tmp);
    return {};
}



std::string gauss5_launch_stream(const uint8_t* d_in, uint8_t* d_out, uint16_t* d_tmp, int w, int h, hipStream_t stream)
{
    if (!d_in || !d_out || !d_tmp || w <= 0 || h <= 0) return "gauss5_launch_stream: bad args";
    constexpr int BW = 16, BH = 16;
    dim3 block(BW, BH);
    dim3 grid((w + BW - 1) / BW, (h + BH - 1) / BH);
    hpass_gauss5<<<grid, block, 0, stream>>>(d_in, d_tmp, w, h);
    auto st = hipGetLastError();
    if (st != hipSuccess) return std::string("gauss hpass launch failed: ") + hipGetErrorString(st);
    vpass_gauss5<<<grid, block, 0, stream>>>(d_tmp, d_out, w, h);
    st = hipGetLastError();
    if (st != hipSuccess) return std::string("gauss vpass launch failed: ") + hipGetErrorString(st);
    return {};
}